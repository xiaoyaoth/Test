#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"
#include "thrust\scan.h"
#include "thrust\sort.h"
#include "thrust\device_ptr.h"
#include "thrust\device_vector.h"
#include <stdio.h>

#define AGENTNO 1024
#define BUFFERSIZE 2048
#define BLOCK_SIZE 128
#define DICE 0.9
#define VERBOSE 0

#define checkCudaErrors(err)	__checkCudaErrors(err, __FILE__, __LINE__)
inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
{
	if( hipSuccess != err) {
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n", file, line, (int)err, hipGetErrorString( err ) );
		exit(-1);
	}
}

#define getLastCudaError(msg)	__getLastCudaError (msg, __FILE__, __LINE__)
inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n", file, line, errorMessage, (int)err, hipGetErrorString( err ) );
		system("PAUSE");
		exit(-1);
	}
}

__device__ unsigned int numAg;
__device__ unsigned int numAgTemp;
int numAg_h;

class GRandom;
class GAgent;
class GModel;

class GRandom {
	hiprandState rState;
public:
	__device__ GRandom(int seed, int idx){
		hiprand_init(seed, idx, 0, &rState);
	}
	__device__ float genUniform(){
		return hiprand_uniform(&rState);
	}
};

class GModel {
	GAgent **alist;
};

class GAgent{
public:
	int id;
	GRandom *random;
	__device__ GAgent(){
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		this->random = new GRandom(2345, idx);
	}
	__device__ virtual void step(GModel *gm) = 0;
};

class AgentOne : public GAgent {
	__device__ void step(GModel *gm) {
		printf("%d ", this->id);
	}
};

class AgentTwo : public GAgent {
	__device__ void step(GModel *gm) {
	}
};

__global__ void init(GAgent **alist){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	AgentOne *ag = new AgentOne();
	ag->id = idx;
	alist[idx] = ag;
}

__device__ int atomicAdd1(int* address, int val)
{
	unsigned int old = *address, assumed;
	do {
		assumed = old;
		old = atomicCAS(address, assumed, (val + assumed)); 
	} while (assumed != old);
	return old;
}

__global__ void setNumAg(){
	numAg = numAgTemp;
}

__global__ void insert(GAgent **alist, GModel *gm){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	GAgent *ag = alist[idx];
	float dice = ag->random->genUniform();
	ag->step(gm);
	if (dice < DICE) {
		AgentOne *newAg = new AgentOne();
		int newIdx = atomicInc((unsigned int *)&numAg, BUFFERSIZE);
		newAg->id = newIdx;
		alist[newIdx] = newAg;
	}
	float test = dice;
}

__global__ void remove(GAgent **alist, GModel *gm){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < numAg /2) {
		atomicDec(&numAgTemp, BUFFERSIZE);
		GAgent *ag = alist[idx];
		delete ag;
		alist[idx] = NULL;
	}
}

__global__ void check(GAgent **alist) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < numAg) {
		GAgent *ag = alist[idx];
		float dice = ag->random->genUniform();
	}
}

struct AListComp {
	//returning true means the one with satisfied condition will be put in the front
	__host__ __device__
		bool operator()(const GAgent *a, const GAgent *b) {
			if (a != NULL)
				return true;
			return false;
	}
};

void sortAList(GAgent **alist){
	thrust::device_ptr<GAgent *> alist_ptr(alist);
	typedef thrust::device_vector<GAgent *>::iterator Iter;
	Iter key_begin(alist_ptr);
	Iter key_end(alist_ptr + numAg_h);
	thrust::sort(key_begin, key_end, AListComp());
	getLastCudaError("sort_hash_kernel");
}

int main()
{
	AgentOne *ag = new AgentOne();
	ag->id = 100;
	delete ag;
	ag->id = 0;

	numAg_h = AGENTNO;
	printf("size of hiprandState: %d\n", sizeof(hiprandState));
	int GRID_SIZE = (int)(AGENTNO/BLOCK_SIZE);
	hipMemcpyToSymbol(HIP_SYMBOL(numAgTemp), &numAg_h, sizeof(int), 0, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	GAgent **a_dev;
	hipMalloc((void**)&a_dev, BUFFERSIZE*sizeof(GAgent*));
	hipMemset(a_dev, 0, BUFFERSIZE*sizeof(GAgent*));

	GModel *gm_dev;
	hipMalloc((void**)&gm_dev, sizeof(GModel));

	init<<<GRID_SIZE, BLOCK_SIZE>>>(a_dev);

	hipEventRecord(start, 0);
	//insert<<<GRID_SIZE, BLOCK_SIZE>>>(a_dev, gm_dev);
	for (int i = 0; i < 10; i++) {
		setNumAg<<<1, 1>>>();
		remove<<<GRID_SIZE, BLOCK_SIZE>>>(a_dev, gm_dev);
		sortAList(a_dev);
		hipMemcpyFromSymbol(&numAg_h, HIP_SYMBOL(numAgTemp), sizeof(int), 0, hipMemcpyDeviceToHost);
		GRID_SIZE = numAg_h%BLOCK_SIZE==0 ? numAg_h/BLOCK_SIZE : numAg_h/BLOCK_SIZE + 1;
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float time = 1.0;
	hipEventElapsedTime(&time, start, stop);
	printf("time: %f\n", time);
	hipMemcpyFromSymbol(&numAg_h, HIP_SYMBOL(numAg), sizeof(int), 0, hipMemcpyDeviceToHost);
	printf("numAg: %d\n", numAg_h);

	//GRID_SIZE = (int)(BUFFERSIZE/BLOCK_SIZE);
	//check<<<GRID_SIZE, BLOCK_SIZE>>>(a_dev);

	system("PAUSE");
	return 0;
}

__global__ void scanInit(int *a_dev){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	a_dev[idx] = 1;
}

__global__ void scanInsert(){
	AgentOne *ag = new AgentOne();
}

int main2(){
	int *a_dev;
	checkCudaErrors(hipMalloc((void**)&a_dev, AGENTNO * sizeof(int)));
	thrust::device_ptr<int> a_ptr(a_dev);
	thrust::device_vector<int>::iterator key_begin(a_ptr);
	thrust::device_vector<int>::iterator key_end(a_ptr + AGENTNO);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int GRID_SIZE = AGENTNO/BLOCK_SIZE;
	int SMEM_SIZE = BLOCK_SIZE * sizeof(int);
	scanInit<<<GRID_SIZE, BLOCK_SIZE>>>(a_dev);

	hipEventRecord(start, 0);
	//scanInsert<<<GRID_SIZE, BLOCK_SIZE>>>();
	thrust::sort(key_begin, key_end);
	thrust::inclusive_scan(key_begin, key_end, a_ptr);
	hipEventRecord(stop, 0);  
	hipEventSynchronize(stop);

	float insertTime = 0;
	hipEventElapsedTime(&insertTime, start, stop);
	printf("insert time: %f\n", insertTime);

	int *a_host = (int*)malloc(AGENTNO * sizeof(int));
	hipMemcpy(a_host, a_dev, sizeof(int) * AGENTNO, hipMemcpyDeviceToHost);
	printf("%d ", a_host[AGENTNO-1]);
	system("PAUSE");
	return 0;
}